#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>

#define NUM_THREADS 256

// Put any static global variables here that you will use throughout the simulation.
int blks;

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    // r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

__global__ void compute_forces_gpu(particle_t* particles, int num_parts) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particles[tid].ax = particles[tid].ay = 0;
    for (int j = 0; j < num_parts; j++)
        apply_force_gpu(particles[tid], particles[j]);
}

__global__ void move_gpu(particle_t* particles, int num_parts, double size) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particle_t* p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //
    //  bounce from walls
    //
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}

void init_simulation(particle_t* parts, int num_parts, double size) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here

    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;

    /////
    num_bins_side = int(size / cutoff);
    num_bins = num_bins_side * num_bins_side;
    size_bin = size / num_bins_side;
    /////
}

__global__ void create_bin_counts(particle_t* parts, int num_parts, int* bin_counts, int size_bin, int num_bins) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) {
      return;
    }
    // particle_t& part = parts[tid];

    // int bin_x = int(part.x / size_bin);
    // int bin_y = int(part.y / size_bin);
    // int bin_num = bin_x + bin_y * num_bins;

    int bin_x = int(parts[tid].x / size_bin);
    int bin_y = int(parts[tid].y / size_bin);
    int bin_num = bin_x + bin_y * num_bins;

    // int* cpu_bin_num = (int*) malloc(sizeof(int));
    // hipMemcpy(cpu_bin_num, bin_num, sizeof(int), hipMemcpyDeviceToHost);

    // std::cout << cpu_bin_num << ",\t";

    bin_counts[bin_num] = 5;

    // atomicAdd(&bin_counts[bin_num], 1);
}

void simulate_one_step(particle_t* parts, int num_parts, double size) {
    // parts live in GPU memory
    // Rewrite this function

    /////
    if (!parts_ordered_inds) {
        hipMalloc((void**) &parts_ordered_inds, size * sizeof(int));
    }

    if (!bin_counts) {
        hipMalloc((void**) &bin_counts, num_bins * sizeof(int));
    }

    hipMemset(parts_ordered_inds, -1, size * sizeof(int));
    hipMemset(bin_counts, 0, num_bins * sizeof(int));

    int* cpu_bin_counts = (int*) malloc(num_bins * sizeof(int));
    hipMemcpy(cpu_bin_counts, bin_counts, num_bins * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 2; i++) {
        std::cout << i << ": "<<  cpu_bin_counts[i] << ",\t";
    }

    create_bin_counts<<<blks, NUM_THREADS>>>(parts, num_parts, bin_counts, size_bin, num_bins);
    /////

    // Compute forces
    compute_forces_gpu<<<blks, NUM_THREADS>>>(parts, num_parts);

    // Move particles
    move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, size);
}
